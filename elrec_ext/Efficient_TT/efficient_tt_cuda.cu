#include "hip/hip_runtime.h"

#include <assert.h>
#include <ATen/ATen.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
// #include <ATen/CUDAGeneratorImpl.h>
#include <ATen/TensorUtils.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "c10/cuda/CUDAStream.h"
#include "c10/util/Exception.h"
#include "hashtbl_cuda_utils.cuh"
#include "tt_cuda_utils.cuh"
#include <iostream>

// #include "cub-1.8.0/cub/device/device_radix_sort.cuh"

#define WARP_SIZE 32
#define eps 1e-5
#define MAX_BATCH_SIZE 8192

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)


#define CHECK_CUDA(x)                                                          \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)                                                    \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)                                                         \
  CHECK_CUDA(x);                                                               \
  CHECK_CONTIGUOUS(x)


using namespace at;

// float** group_map; // content point to result of intermediate result.
// int32_t* group_flag; // indicate group_map is/not have update
// int32_t* group_idx;
// float *cache;
// float *output_d;
// int32_t *group_idx_h;


void init_cuda(
    int32_t device_id,
    const std::vector<int>& tt_p_shape,
    const std::vector<int>& tt_q_shape,
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    int32_t batch_size,
    int32_t feature_dim
)
{
    // hipSetDevice(device_id);
    // int device_id_;
    // hipGetDevice(&device_id_);
    // std::cout << "device_id: " << device_id_ << std::endl;
    int min_length = tt_p_shape[1] * tt_p_shape[2];
    return;

//     if(!group_map)
//     {
//       // int min_length = 370 * 370;
//       int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];

//       hipMalloc(&group_map, min_length*sizeof(float*));
//       hipMalloc(&group_flag, min_length*sizeof(int32_t);
//      // hipMalloc(&group_idx, sizeof(int32_t));  // GPU Mem
//       hipMallocManaged(&group_idx, sizeof(int32_t));  // unified Mem
//       hipMalloc(&cache, min_length * cache_dim * sizeof(float));
//       hipMalloc(&output_d, batch_size * feature_dim * sizeof(float));

//       printf("Malloced GPU Mem: %ld\n",min_length * cache_dim * sizeof(float));
//     }
}

void check_init(
    int32_t device_id,
    const std::vector<int>& tt_p_shape,
    const std::vector<int>& tt_q_shape,
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    int32_t batch_size,
    int32_t feature_dim,
    at::Tensor tensor_group_map,
    at::Tensor tensor_group_flag,
    at::Tensor tensor_group_idx,
    at::Tensor tensor_cache
) {
  int min_length = tt_p_shape[0] * tt_p_shape[1];
  int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];

  const std::string error_string = std::string("Eff TT init check failed: ");

  auto bytes = [](at::Tensor t) -> int64_t {
    return t.element_size() * t.numel();
  };

  TORCH_CHECK(min_length*sizeof(float*) ==  bytes(tensor_group_map), error_string + "group_map");
  TORCH_CHECK(min_length*sizeof(int32_t) == bytes(tensor_group_flag), error_string + "group_flag");
  TORCH_CHECK(sizeof(int32_t) == bytes(tensor_group_idx), error_string + "group_idx");
  TORCH_CHECK(min_length * cache_dim * sizeof(float) == bytes(tensor_cache), error_string + "cache");
  // float** group_map = (float**)tensor_group_map.data_ptr();
  // int32_t* group_flag = (int32_t*)tensor_group_flag.data_ptr();
  // int32_t* group_idx = (int32_t*)tensor_group_idx.data_ptr();
  // float * ache = (float*)tensor_cache.data_ptr();
  // float * output_d = (float*)tensor_output_d.data_ptr();
}

inline void cuda_gemm_batched_fp32_fp32(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float* alpha,
    void** a_array,
    int lda,
    void** b_array,
    int ldb,
    float* beta,
    void** c_array,
    int ldc,
    int batch_count) 
{
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(handle, c10::cuda::getCurrentCUDAStream());

  hipblasGemmBatchedEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      a_array,
      HIP_R_32F,
      lda,
      b_array,
      HIP_R_32F,
      ldb,
      beta,
      c_array,
      HIP_R_32F,
      ldc,
      batch_count,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

inline void stream_cuda_gemm_batched_fp32_fp32(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    float* alpha,
    void** a_array,
    int lda,
    void** b_array,
    int ldb,
    float* beta,
    void** c_array,
    int ldc,
    int batch_count,
    hipStream_t stream) 
{
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(handle, stream);
  hipblasGemmBatchedEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      a_array,
      HIP_R_32F,
      lda,
      b_array,
      HIP_R_32F,
      ldb,
      beta,
      c_array,
      HIP_R_32F,
      ldc,
      batch_count,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

__global__ void prepare_batch_gemm_pointers_3_core(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    int32_t index_length,
    int32_t output_length,
    int32_t cache_length,
    int32_t cache_dim,
    const int64_t* index,
    const int64_t* tt_p_shape,
    const int64_t* tt_q_shape,
    const int64_t* tt_ranks,
    float* tt_core_0,
    float* tt_core_1,
    float* tt_core_2,
    float* cache,
    float* result,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** group_map,
    int32_t* group_flag,
    int32_t* group_idx
    )
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(n >= batch_size)
    return;

  int idx_start = n;
  int idx_end = n + 1;

  int group;
  for(int i=idx_start;i<idx_end;i++){
    int idx = *(index + i);
    // float tmp = float(idx)/tt_p_shape[2];
    // group = floor(tmp);
    group = idx / tt_p_shape[2];
    int I3 = idx % tt_p_shape[2];
    if(atomicCAS(group_flag + group, 0, 1)==0)
    {
      int cache_idx = atomicAdd(group_idx, 1);
      // int I1 = floor(float(group)/tt_p_shape[1]);
      int I1 = idx / (tt_p_shape[1] * tt_p_shape[2]);
      int I2 = group%tt_p_shape[1];
      // int I2 = floor(float(group)/tt_p_shape[0]);
      // int I1 = group%tt_p_shape[0];

      a_ptr[cache_idx] = tt_core_1 + I2 * tt_ranks[1] * tt_q_shape[1] * tt_ranks[2]; 
      b_ptr[cache_idx] = tt_core_0 + I1 * tt_q_shape[0] * tt_ranks[1];
      c_ptr[cache_idx] = cache + cache_idx * cache_dim;
      group_map[group] = cache + cache_idx * cache_dim;
    }
    a_ptr[cache_length + n] = tt_core_2 + I3 * (tt_q_shape[2] * tt_ranks[2]); 
    b_ptr[cache_length + n] = group_map[group]; // from cache
    c_ptr[cache_length + n] = result + n * output_length;
  }
}


__global__ void update_group_map(
  int32_t batch_size,
  int32_t cache_length,
  int32_t index_length,
  const int64_t* index,
  const int64_t* tt_p_shape,
  float** group_map,
  float** __restrict__ b_ptr
)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if(n >= batch_size)
    return;
  
  int idx = *(index + n);
  // float tmp = float(idx)/tt_p_shape[2];
  // int group = floor(tmp);
  int group = idx / tt_p_shape[2];
  // if(!b_ptr[cache_length + n])
  {
    b_ptr[cache_length + n] = group_map[group];
    // printf("%p\n",b_ptr[cache_length + n]);
  }
}

Tensor Efficient_TT_forward_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    const Tensor index,
    const std::vector<int>& tt_p_shape, //[i1,i2,i3]
    const std::vector<int>& tt_q_shape, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shape, //[i1,i2,i3]
    const Tensor tensor_q_shape, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    const std::vector<Tensor>& tt_cores,
    at::Tensor tensor_group_map,
    at::Tensor tensor_group_flag,
    at::Tensor tensor_group_idx,
    at::Tensor tensor_cache,
    at::Tensor tensor_batch_ratio
){
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(tt_cores[0].get_device());
    // auto output =
    //   at::zeros({batch_size, feature_dim}, tt_cores[0].options().dtype(at::kFloat));
    // auto output =
      // at::zeros({batch_size, feature_dim}, tt_cores[0].options().dtype(at::kFloat));
    TORCH_WARN("WARNING! 'Efficient_TT_forward_cuda' is not safe! Please use 'Efficient_TT_forward_bag_cuda'!");
    
    tensor_group_flag.zero_();
    tensor_group_idx.zero_();

    float** group_map = (float**)tensor_group_map.data_ptr();
    int32_t* group_flag = (int32_t*)tensor_group_flag.data_ptr();
    int32_t* group_idx = (int32_t*)tensor_group_idx.data_ptr();
    float * cache = (float*)tensor_cache.data_ptr();
    // float * output_d = (float*)tensor_output_d.data_ptr();

    auto output = at::zeros({batch_size, feature_dim}, tt_cores[0].options());
    float *output_d = output.data_ptr<float>();
    
    int32_t index_length = index.sizes()[0];
    int32_t num_core = tt_p_shape.size();
    int32_t num_rank = tt_p_shape.size() + 1;
    // int32_t cache_length = tt_p_shape[1] * tt_p_shape[2];
    int32_t cache_length = tt_p_shape[0] * tt_p_shape[1];
    int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];
    int32_t output_length = tt_q_shape[0] * tt_q_shape[1] * tt_q_shape[2];

    // // printf("\ncache_dim:%d,cache_length:%d,num_core:%d,num_rank:%d\n",cache_dim,cache_length,num_core,num_rank);

    auto a_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();

    if(!group_map)
    {
      TORCH_CHECK(false, "Try to init cuda again");
    }

    // hipMemset(group_flag, 0, cache_length*sizeof(int32_t)); //set to zero
    // hipMemset(group_idx, 0, sizeof(int32_t)); //set to zero

    // int32_t threads = 256;
    int32_t threads = (batch_size > 256 ? 256 : 32);
    // int32_t num_blocks = (table_length + threads - 1) / threads; 
    int32_t num_blocks = (batch_size + threads - 1) / threads;
    prepare_batch_gemm_pointers_3_core<<<num_blocks, threads>>>(  // one thread lookup one row
      batch_size,
      table_length,
      feature_dim,
      index_length,
      output_length,
      cache_length,
      cache_dim,

      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      (const int64_t*)tensor_q_shape.data_ptr(),
      (const int64_t*)tensor_ranks.data_ptr(),
      (float*)tt_cores[0].data_ptr(),
      (float*)tt_cores[1].data_ptr(),
      (float*)tt_cores[2].data_ptr(),
      cache,
      output_d,
      a_ptr,
      b_ptr,
      c_ptr,
      group_map,
      group_flag,
      group_idx
    );

    // use cuBlas batched gemm compute cache
    float alpha = 1.0;
    float beta = 0.0;

    int batch_cnt = tensor_group_idx.item().to<int>();
    // hipMemcpy(&batch_cnt, group_idx, sizeof(int32_t), hipMemcpyDeviceToHost);
    tensor_batch_ratio.fill_((float) batch_size / batch_cnt);
    // std::cout << "batch_cnt: " << batch_cnt << "tensor_batch_ratio: " << tensor_batch_ratio << std::endl;

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[1]*tt_ranks[2], //n
      tt_q_shape[0],//m
      tt_ranks[1],//k
      &alpha,
      (void**)a_ptr,
      tt_q_shape[1]*tt_ranks[2], //n
      (void**)b_ptr,
      tt_ranks[1], // k
      &beta,
      (void**)c_ptr,
      tt_q_shape[1]*tt_ranks[2], // n
      batch_cnt
    );

    update_group_map<<<num_blocks, threads>>>(
      batch_size,
      cache_length,
      index_length,
      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      group_map,
      b_ptr
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[2], //n
      tt_q_shape[0] * tt_q_shape[1],//m
      tt_ranks[2],//k
      &alpha,
      (void**)(a_ptr+cache_length),
      tt_q_shape[2], //n
      (void**)(b_ptr+cache_length),
      tt_ranks[2], // k
      &beta,
      (void**)(c_ptr+cache_length),
      tt_q_shape[2], // n
      batch_size
    );

    // hipMemcpy(
      // output.data_ptr();
      // output_d, 
    //   batch_size * feature_dim * sizeof(float), 
    //   hipMemcpyDeviceToHost /// Are you sure????
    // );

    return output;
}

__global__ void reduce_output_kernel(
    int32_t N,  // batch cnt
    int32_t B,  // batch size
    int32_t D,  // feature dim
    int32_t output_dim, // output_dim
    const int64_t* __restrict__ rowidx,
    const float* __restrict__ tr_last,
    float* __restrict__ output) {
  int32_t indice_id = blockIdx.x * blockDim.y + threadIdx.y;
  if (indice_id >= N) {
    // don't have *warp* divergence since we launch full warps in blockDim.x,
    // so we can just exit this warp entirely.
    return;
  }
  // check if this warp is responsible for this whole segment.
  bool segment_start =
      (indice_id == 0 || rowidx[indice_id - 1] != rowidx[indice_id]);
  if (!segment_start) {
    // don't have *warp* divergence since we launch full warps in blockDim.x,
    // so we can just exit this warp entirely.
    return;
  }
  int64_t row_index = rowidx[indice_id];
  // now, find the end of the segment (and thus the segment length `SL`).
  int32_t SL = 1;
  while (indice_id + SL < N && rowidx[indice_id + SL] == row_index) {
    SL += 1;
  }
  for (int32_t d = threadIdx.x; d * 4 < D; d += blockDim.x) {
    Vec4T<float> sum(&output[row_index * D + d * 4]);
    for (int32_t sl = 0; sl < SL; ++sl) {
      Vec4T<float> tr(&tr_last[(indice_id + sl) * output_dim + d * 4]);
      sum.acc.x += tr.acc.x;
      sum.acc.y += tr.acc.y;
      sum.acc.z += tr.acc.z;
      sum.acc.w += tr.acc.w;
    }
    sum.store(&output[row_index * D + d * 4]);
  }
}



Tensor Efficient_TT_forward_bag_cuda(
    int32_t batch_cnt,
    int32_t table_length,
    int32_t feature_dim,
    Tensor index,
    Tensor offsets,
    Tensor rowidx,
    const std::vector<int>& tt_p_shape, //[i1,i2,i3]
    const std::vector<int>& tt_q_shape, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    Tensor tensor_p_shape, //[i1,i2,i3]
    Tensor tensor_q_shape, //[j1,j2,j3]
    Tensor tensor_ranks, //[1,r1,r2,1]
    const std::vector<Tensor>& tt_cores,
    at::Tensor tensor_group_map,
    at::Tensor tensor_group_flag,
    at::Tensor tensor_group_idx,
    at::Tensor tensor_cache,
    at::Tensor tensor_batch_ratio
){
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(tt_cores[0].get_device());
    // auto output =
    //   at::zeros({batch_size, feature_dim}, tt_cores[0].options().dtype(at::kFloat)
    // auto output =
      // at::zeros({batch_size, feature_dim}, tt_cores[0].options().dtype(at::kFloat));

    int32_t batch_size = index.numel();   // real batch cnt
    TORCH_CHECK(batch_size == batch_cnt);
    int32_t batch_sz = offsets.numel() - 1;  // batch size
    
    tensor_group_flag.zero_();
    tensor_group_idx.zero_();

    float** group_map = (float**)tensor_group_map.data_ptr();
    int32_t* group_flag = (int32_t*)tensor_group_flag.data_ptr();
    int32_t* group_idx = (int32_t*)tensor_group_idx.data_ptr();
    float * cache = (float*)tensor_cache.data_ptr();
    // float * output_d = (float*)tensor_output_d.data_ptr();

    
    int32_t index_length = index.sizes()[0];
    int32_t num_core = tt_p_shape.size();
    int32_t num_rank = tt_p_shape.size() + 1;
    // int32_t cache_length = tt_p_shape[1] * tt_p_shape[2];
    int32_t cache_length = tt_p_shape[0] * tt_p_shape[1];
    int32_t cache_dim = tt_q_shape[0] * tt_q_shape[1] * tt_ranks[2];
    int32_t output_length = tt_q_shape[0] * tt_q_shape[1] * tt_q_shape[2];
    

    auto output = at::zeros({batch_size, output_length}, tt_cores[0].options());
    auto reduced_output = at::zeros({batch_sz, feature_dim}, tt_cores[0].options());
    float *output_d = output.data_ptr<float>();

    // // printf("\ncache_dim:%d,cache_length:%d,num_core:%d,num_rank:%d\n",cache_dim,cache_length,num_core,num_rank);

    auto a_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
      {cache_length + batch_size}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();

    if(!group_map)
    {
      TORCH_CHECK(false, "Try to init cuda again");
    }

    // hipMemset(group_flag, 0, cache_length*sizeof(int32_t)); //set to zero
    // hipMemset(group_idx, 0, sizeof(int32_t)); //set to zero

    // int32_t threads = 256;
    int32_t threads = (batch_size > 256 ? 256 : 32);
    int32_t num_blocks = (batch_size + threads - 1) / threads;
    prepare_batch_gemm_pointers_3_core<<<
      num_blocks, threads,0, c10::cuda::getCurrentCUDAStream()>>>(  // one thread lookup one row
      batch_size,
      table_length,
      feature_dim,
      index_length,
      output_length,
      cache_length,
      cache_dim,

      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      (const int64_t*)tensor_q_shape.data_ptr(),
      (const int64_t*)tensor_ranks.data_ptr(),
      (float*)tt_cores[0].data_ptr(),
      (float*)tt_cores[1].data_ptr(),
      (float*)tt_cores[2].data_ptr(),
      cache,
      output_d,
      a_ptr,
      b_ptr,
      c_ptr,
      group_map,
      group_flag,
      group_idx
    );
    // int32_t *group_idx_h = (int32_t*)malloc(sizeof(int32_t));
    // hipMemcpy(group_idx_h, group_idx, sizeof(int32_t), hipMemcpyDeviceToHost);
    // printf("final group idx:%d\n",*group_idx_h);

    // use cuBlas batched gemm compute cache
    float alpha = 1.0;
    float beta = 0.0;

    int forward_gemm_batch_cnt = tensor_group_idx.item().to<int>();
    // hipMemcpy(&batch_cnt, group_idx, sizeof(int32_t), hipMemcpyDeviceToHost);
    tensor_batch_ratio.fill_((float) batch_size / forward_gemm_batch_cnt);

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[1]*tt_ranks[2], //n
      tt_q_shape[0],//m
      tt_ranks[1],//k
      &alpha,
      (void**)a_ptr,
      tt_q_shape[1]*tt_ranks[2], //n
      (void**)b_ptr,
      tt_ranks[1], // k
      &beta,
      (void**)c_ptr,
      tt_q_shape[1]*tt_ranks[2], // n
      forward_gemm_batch_cnt
    );

    update_group_map<<<num_blocks, threads>>>(
      batch_size,
      cache_length,
      index_length,
      (const int64_t*)index.data_ptr(),
      (const int64_t*)tensor_p_shape.data_ptr(),
      group_map,
      b_ptr
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      tt_q_shape[2], //n
      tt_q_shape[0] * tt_q_shape[1],//m
      tt_ranks[2],//k
      &alpha,
      (void**)(a_ptr+cache_length),
      tt_q_shape[2], //n
      (void**)(b_ptr+cache_length),
      tt_ranks[2], // k
      &beta,
      (void**)(c_ptr+cache_length),
      tt_q_shape[2], // n
      batch_size
    );

    // hipMemcpy(
      // output.data_ptr();
      // output_d, 
    //   batch_size * feature_dim * sizeof(float), 
    //   hipMemcpyDeviceToHost /// Are you sure????
    // );
    int32_t tx = kWarpSize;
    int32_t ty = 1024 / tx;
    dim3  thds(tx, ty);
    num_blocks = (batch_size + ty - 1) / ty;
    reduce_output_kernel<<<num_blocks,
      thds,
      0,
      c10::cuda::getCurrentCUDAStream()>>>(
        batch_size,
        batch_sz,
        feature_dim,
        output_length,
        rowidx.data_ptr<int64_t>(),
        output.data_ptr<float>(),
        reduced_output.data_ptr<float>()
    );

    return reduced_output;
}


__global__ void compute_rowidx_kernel(
    int32_t B,
    const int64_t* __restrict__ offsets,
    int64_t* __restrict__ rowidx) {
  int32_t b = blockIdx.x * blockDim.y + threadIdx.y;
  // printf("b < B:%d\n", b < B);
  if (b < B) {
    int64_t colidx_start = offsets[b];
    int64_t colidx_end = offsets[b + 1];
    int32_t L = colidx_end - colidx_start;
    for (int32_t l = threadIdx.x; l < L; l += blockDim.x) {
      rowidx[l + colidx_start] = b;
    }
  }
}

at::Tensor compute_rowidx_cuda(
    Tensor indices,
    Tensor offsets
) {
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(indices.get_device());

  CHECK_INPUT(indices);
  CHECK_INPUT(offsets);

  TORCH_CHECK(indices.dtype() == at::kLong)
  auto rowidx = at::empty_like(indices);

  int32_t B = offsets.numel() - 1;

  int32_t tx = 8;
  int32_t ty = 32;
  compute_rowidx_kernel<<<
      div_round_up(B, ty),
      dim3(tx, ty),
      0,
      c10::cuda::getCurrentCUDAStream()>>>(
        B,
        offsets.data_ptr<int64_t>(),
        rowidx.data_ptr<int64_t>()
      );
  
  return rowidx;
}


//=============================================================================================================================
__global__ void prepare_batch_gemm_pointers_3_core_backward(
    int32_t N,
    const int64_t* __restrict__ index,
    const int64_t* tt_p_shape,

    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_1,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_2,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_1,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_2,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_output,
    int32_t* __restrict__ tt_idx,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** __restrict__ a0_ptr,
    float** __restrict__ b0_ptr,
    float** __restrict__ c0_ptr,
    float** __restrict__ a1_ptr,
    float** __restrict__ b1_ptr,
    float** __restrict__ c1_ptr) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < N) {
    int32_t idx = *(index+n);

    // float tmp = float(idx)/tt_p_shape[2];
    // int group = floor(tmp);
    // int I3 = idx % tt_p_shape[2];
    // int I1 = floor(float(group)/tt_p_shape[1]);
    // int I2 = group%tt_p_shape[1];
    int group = idx / tt_p_shape[2];
    int I3 = idx % tt_p_shape[2];
    int I1 = idx / (tt_p_shape[1] * tt_p_shape[2]);
    int I2 = group%tt_p_shape[1];

    tt_idx[0 * N + n] = I1;
    tt_idx[1 * N + n] = I2;
    tt_idx[2 * N + n] = I3;
    float* tr_0_ptr = (float*)&(tr_0[n][0]);
    float* d_output_ptr = (float*)&(d_output[n][0]);
    float* tt_cores_0_ptr = (float*)&(tt_cores_0[I1][0]);
    float* tt_cores_1_ptr = (float*)&(tt_cores_1[I2][0]);
    a_ptr[0 * N + n] = tt_cores_1_ptr;
    b_ptr[0 * N + n] = tt_cores_0_ptr;
    c_ptr[0 * N + n] = tr_0_ptr;

    a0_ptr[1 * N + n] = tr_0_ptr;
    b0_ptr[1 * N + n] = d_output_ptr;
    c0_ptr[1 * N + n] = (float*)&(tr_tt_cores_2[n][0]);
    a1_ptr[1 * N + n] = d_output_ptr;
    b1_ptr[1 * N + n] = (float*)&(tt_cores_2[I3][0]);
    c1_ptr[1 * N + n] = tr_0_ptr;

    a0_ptr[0 * N + n] = tt_cores_0_ptr;
    b0_ptr[0 * N + n] = tr_0_ptr;
    c0_ptr[0 * N + n] = (float*)&(tr_tt_cores_1[n][0]);
    a1_ptr[0 * N + n] = tr_0_ptr;
    b1_ptr[0 * N + n] = tt_cores_1_ptr;
    c1_ptr[0 * N + n] = (float*)&(tr_tt_cores_0[n][0]);
  }
}





__global__ void update_d_tt_cores_kernel(
    int32_t N,
    int32_t D,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_tt_cores) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n < N) {
    auto idx = __ldg(&tt_idx[n]);
    for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
      atomicAdd(&(d_tt_cores[idx][d]), 0.1 * tr_tt_cores[n][d]);
    }
  }
}

__global__ void update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_tt_cores,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t b = blockIdx.x * blockDim.y + threadIdx.y;
  if (b >= B) {
    return;
  }
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    // tt_core[b][d] -= learning_rate * d_tt_cores[b][d];
    tt_core[b][d] -= d_tt_cores[b][d];
  }
}

__global__ void fused_update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= B) {
    return;
  }
  learning_rate *= -1;
  auto idx = __ldg(&tt_idx[n]);
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    auto delta = learning_rate * tr_tt_cores[n][d];
    atomicAdd(&(tt_core[idx][d]), delta);
  }
}

void Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;

    std::vector<Tensor> d_tt_cores;
    std::vector<Tensor> tr_tt_cores;
    d_tt_cores.push_back(at::zeros_like(tt_cores[0]));
    d_tt_cores.push_back(at::zeros_like(tt_cores[1]));
    d_tt_cores.push_back(at::zeros_like(tt_cores[2]));
    
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({batch_count, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * batch_count}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    int32_t start_idx = 0;
    int32_t end_idx = start_idx + batch_count;
    int32_t N = end_idx - start_idx;

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        N,
        (const int64_t*)index.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        d_output.packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        N);
    // for (int32_t t = 0; t < T - 2; ++t)
    // backward propagation

    for (int32_t t = T - 2; t >= 0; --t) {
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            n[t],
            k[t],
            m[t],
            &alpha,
            (void**)&(b0_ptr[t * N]),
            n[t],
            (void**)&(a0_ptr[t * N]),
            k[t],
            &beta,
            (void**)&(c0_ptr[t * N]),
            n[t],
            N);
        int32_t D_0 = tt_cores[t + 1].size(1);
        int32_t tx_0 = std::min(1024, D_0);
        int32_t ty_0 = 1024 / tx_0;
        update_d_tt_cores_kernel<<<
            div_round_up(N, ty_0),
            dim3(tx_0, ty_0),
            0,
            c10::cuda::getCurrentCUDAStream()>>>(
            N,
            D_0,
            &(tt_idx.data_ptr<int32_t>()[(t + 1) * N]),
            tr_tt_cores[t + 1].packed_accessor64<float, 2, RestrictPtrTraits>(),
            d_tt_cores[t + 1].packed_accessor64<float, 2, RestrictPtrTraits>());
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            k[t],
            m[t],
            n[t],
            &alpha,
            (void**)&(b1_ptr[t * N]),
            n[t],
            (void**)&(a1_ptr[t * N]),
            n[t],
            &beta,
            (void**)&(c1_ptr[t * N]),
            k[t],
            N);
        if (t == 0) {
            int32_t D_1 = tt_cores[0].size(1);
            int32_t tx_1 = std::min(1024, D_1);
            int32_t ty_1 = 1024 / tx_1;
            update_d_tt_cores_kernel<<<
                div_round_up(N, ty_1),
                dim3(tx_1, ty_1),
                0,
                c10::cuda::getCurrentCUDAStream()>>>(
                N,
                D_1,
                &(tt_idx.data_ptr<int32_t>()[t * N]),
                tr_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
                d_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>());
        }
    } // for (int32_t t = T - 2; t >=0 ; --t)

    for (int32_t t = 0; t < T; ++t) {
        int32_t y_size = tt_cores[t].size(0);
        int32_t x_size = tt_cores[t].size(1);
        int32_t tx = std::min(1024, y_size);
        int32_t ty = 1024 / tx;
      
        update_tt_cores_sgd_kernel<<<
        div_round_up(x_size, ty),
        dim3(tx, ty),
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
          y_size,
          x_size,
          learning_rate, // hard code
          d_tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>(),
          tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>()
        );
    }
    
  return;
}


void Fused_Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;

    std::vector<Tensor> tr_tt_cores;
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({batch_size, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({batch_count, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * batch_count}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * batch_count}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    int32_t start_idx = 0;
    int32_t end_idx = start_idx + batch_count;
    int32_t N = end_idx - start_idx;

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        N,
        (const int64_t*)index.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        d_output.packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        N);

    for (int32_t t = T - 2; t >= 0; --t) {
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_N,
            HIPBLAS_OP_T,
            n[t],
            k[t],
            m[t],
            &alpha,
            (void**)&(b0_ptr[t * N]),
            n[t],
            (void**)&(a0_ptr[t * N]),
            k[t],
            &beta,
            (void**)&(c0_ptr[t * N]),
            n[t],
            N
            );
        
        cuda_gemm_batched_fp32_fp32(
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            k[t],
            m[t],
            n[t],
            &alpha,
            (void**)&(b1_ptr[t * N]),
            n[t],
            (void**)&(a1_ptr[t * N]),
            n[t],
            &beta,
            (void**)&(c1_ptr[t * N]),
            k[t],
            N
            );
    } // for (int32_t t = T - 2; t >=0 ; --t)

    for (int32_t t = 0; t < T; ++t) {
        int32_t D_0 = tt_cores[t].size(1);
        int32_t tx_0 = std::min(1024, D_0);
        int32_t ty_0 = 1024 / tx_0;
        fused_update_tt_cores_sgd_kernel<<<
        div_round_up(N, ty_0),
        dim3(tx_0, ty_0),
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
          N,
          D_0,
          learning_rate, // hard code
          &(tt_idx.data_ptr<int32_t>()[t * N]),
          tr_tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>(),
          tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>()
        );
    }
    
  return;
}


// Extra_Eff_Fused ============================================================================

__global__ void Extra_Eff_prepare_batch_gemm_pointers_3_core_backward(
    int32_t unique_num,
    const int64_t* __restrict__ unique_index,
    const int64_t* tt_p_shape,

    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_1,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_cores_2,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_1,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores_2,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_0,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_output,
    int32_t* __restrict__ tt_idx,
    float** __restrict__ a_ptr,
    float** __restrict__ b_ptr,
    float** __restrict__ c_ptr,
    float** __restrict__ a0_ptr,
    float** __restrict__ b0_ptr,
    float** __restrict__ c0_ptr,
    float** __restrict__ a1_ptr,
    float** __restrict__ b1_ptr,
    float** __restrict__ c1_ptr) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n < unique_num) {
    int32_t idx = *(unique_index+n);

    // WHY YOU CAST tmp TO FLOAT!!!!?????
    // DONOT YOU KNOW THAT FLOAT OPERATION WILL BRING PRECISION PROBLEM??

    // float tmp = float(idx)/tt_p_shape[2];
    // int group = floor(tmp);
    // int I3 = idx % tt_p_shape[2];
    // int I1 = floor(float(group)/tt_p_shape[1]);
    // int I2 = group%tt_p_shape[1];
    int group = idx / tt_p_shape[2];
    int I3 = idx % tt_p_shape[2];
    int I2 = group % tt_p_shape[1];
    // int I1 = group / tt_p_shape[1];
    int I1 = idx / (tt_p_shape[1] * tt_p_shape[2]);

    tt_idx[0 * unique_num + n] = I1;
    tt_idx[1 * unique_num + n] = I2;
    tt_idx[2 * unique_num + n] = I3;
    float* tr_0_ptr = (float*)&(tr_0[n][0]);
    float* d_output_ptr = (float*)&(d_output[n][0]);
    float* tt_cores_0_ptr = (float*)&(tt_cores_0[I1][0]);
    float* tt_cores_1_ptr = (float*)&(tt_cores_1[I2][0]);
    a_ptr[0 * unique_num + n] = tt_cores_1_ptr; //T1*T2
    b_ptr[0 * unique_num + n] = tt_cores_0_ptr;
    c_ptr[0 * unique_num + n] = tr_0_ptr;

    a0_ptr[1 * unique_num + n] = tr_0_ptr;
    b0_ptr[1 * unique_num + n] = d_output_ptr;
    c0_ptr[1 * unique_num + n] = (float*)&(tr_tt_cores_2[n][0]);
    a1_ptr[1 * unique_num + n] = d_output_ptr;
    b1_ptr[1 * unique_num + n] = (float*)&(tt_cores_2[I3][0]);
    c1_ptr[1 * unique_num + n] = tr_0_ptr;

    a0_ptr[0 * unique_num + n] = tt_cores_0_ptr; 
    b0_ptr[0 * unique_num + n] = tr_0_ptr;
    c0_ptr[0 * unique_num + n] = (float*)&(tr_tt_cores_1[n][0]);
    a1_ptr[0 * unique_num + n] = tr_0_ptr;
    b1_ptr[0 * unique_num + n] = tt_cores_1_ptr;
    c1_ptr[0 * unique_num + n] = (float*)&(tr_tt_cores_0[n][0]);
  }
}


__global__ void compute_unique_gradient(
  int batch_size,
  int feature_dim,
  const int64_t* inverse, 
  PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_input,
  PackedTensorAccessor64<float, 2, RestrictPtrTraits> d_output
)
{
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if(n >= batch_size)
    return;

  int idx = inverse[n];
  for(int i=0;i<feature_dim;i++)
  {
    atomicAdd(&(d_output[idx][i]), d_input[n][i]);
  }
}


__global__ void extra_fused_update_tt_cores_sgd_kernel(
    int32_t B,
    int32_t D,
    float learning_rate,
    const int32_t* __restrict__ tt_idx,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tr_tt_cores,
    PackedTensorAccessor64<float, 2, RestrictPtrTraits> tt_core
    ) {
  int32_t n = blockIdx.x * blockDim.y + threadIdx.y;
  if (n >= B) {
    return;
  }
  auto idx = __ldg(&tt_idx[n]);
  for (int32_t d = threadIdx.x; d < D; d += blockDim.x) {
    auto delta = -learning_rate * tr_tt_cores[n][d];
    atomicAdd(&(tt_core.data()[idx * D + d]), delta);
    // tt_core[idx][d] -= learning_rate * tr_tt_cores[n][d];
  }
}


void Fused_Extra_Efficient_TT_backward_sgd_cuda(
    int32_t batch_size,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,

    const Tensor index,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_output,
    std::vector<Tensor>& tt_cores,
    Tensor sorted_idx,
    Tensor sorted_key
)
{

    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_output.get_device());
    int32_t T = 3;  //3
    int32_t batch_count = batch_size;
    int32_t N = batch_size;
    int32_t unique_num = sorted_idx.size(0);


    auto unique_d = at::zeros({sorted_idx.size(0), d_output.size(1)}, tt_cores[0].options());

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    compute_unique_gradient<<<num_blocks, threads>>>(
      batch_size,
      feature_dim,
      (const int64_t*)sorted_key.data_ptr(),
      d_output.packed_accessor64<float, 2, RestrictPtrTraits>(),
      unique_d.packed_accessor64<float, 2, RestrictPtrTraits>()
    );

    //===================================================================================================================
    std::vector<Tensor> tr_tt_cores;
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[2].size(1)}, tt_cores[2].options()));

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({unique_num, tr_size}, tt_cores[0].options()));
    }

    auto tt_idx =
      at::empty({T * unique_num}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    threads = (unique_num > 256 ? 256 : 32);
    num_blocks = (unique_num + threads - 1) / threads;

    Extra_Eff_prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        (const int64_t*)sorted_idx.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        unique_d.packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        unique_num);

// //======================================================
    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[1],
      k[1],
      m[1],
      &alpha,
      (void**)&(b0_ptr[unique_num]),
      n[1],
      (void**)&(a0_ptr[unique_num]),
      k[1],
      &beta,
      (void**)&(c0_ptr[unique_num]),
      n[1],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[1],
      m[1],
      n[1],
      &alpha,
      (void**)&(b1_ptr[unique_num]),
      n[1],
      (void**)&(a1_ptr[unique_num]),
      n[1],
      &beta,
      (void**)&(c1_ptr[unique_num]),
      k[1],
      unique_num
    );
//=========================================================

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[0],
      k[0],
      m[0],
      &alpha,
      (void**)&(b0_ptr[0]),
      n[0],
      (void**)&(a0_ptr[0]),
      k[0],
      &beta,
      (void**)&(c0_ptr[0]),
      n[0],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[0],
      m[0],
      n[0],
      &alpha,
      (void**)&(b1_ptr[0]),
      n[0],
      (void**)&(a1_ptr[0]),
      n[0],
      &beta,
      (void**)&(c1_ptr[0]),
      k[0],
      unique_num
    );

//=========================================================
  // return (a + b - 1) / b;
  for (int32_t t = 0; t < T; ++t) {
      int32_t D_0 = tt_cores[t].size(1);
      int32_t tx_0 = std::min(1024, D_0);
      int32_t ty_0 = 1024 / tx_0;
      extra_fused_update_tt_cores_sgd_kernel<<<
      div_round_up(unique_num, ty_0),
      dim3(tx_0, ty_0),
      0,
      c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        D_0,
        learning_rate,
        &(tt_idx.data_ptr<int32_t>()[t * unique_num]),
        tr_tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>()
      );
  }
  return;
}

__global__ void generate_d_output_kernel(
  int32_t N,  // batch cnt
  int32_t B, // batch_size
  int32_t D,  // feature dim
  const int64_t* __restrict__ rowidx,
  const float * __restrict__ d_reduced_output,
   float* __restrict__ d_output
) {
  int32_t n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n >= N) {
    return;
  }

  int32_t idx = rowidx[n];
  assert(idx < B);

  for (int32_t d = 0; d < D; ++d) {
    d_output[n * D + d] = d_reduced_output[idx * D + d];
    // atomicAdd(&(d_output[n * D + d]), d_reduced_output[idx * D + d]);
  }
}


at::Tensor aggregate_gradients_v1(
    int32_t N, // batch count
    int32_t B, // batch size
    int32_t feature_dim,
    at::Tensor d_reduced_output,
    at::Tensor rowidx,
    at::Tensor sorted_idx,
    at::Tensor sorted_key) {
  auto options = d_reduced_output.options();

  auto d_output = at::empty({N, feature_dim}, options);
  auto unique_d = at::zeros({sorted_idx.size(0), d_output.size(1)}, options);

  // *************** generate d_output *******************
  int32_t threads = (N > 256 ? 256 : 32);
  int32_t num_blocks = (N + threads - 1) / threads;
  generate_d_output_kernel<<<num_blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
      N,
      B,
      feature_dim,
      rowidx.data_ptr<int64_t>(),
      d_reduced_output.data_ptr<float>(),
      d_output.data_ptr<float>());

  // std::cout << "d_output numel()" << d_output.numel() << std::endl;
  // std::cout << "unique_d numel()" << unique_d.numel() << std::endl;

  // *************** geneate unique_d *******************
  threads = (N > 256 ? 256 : 32);
  num_blocks = (N + threads - 1) / threads;
  compute_unique_gradient<<<num_blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
      N,
      feature_dim,
      (const int64_t*)sorted_key.data_ptr(),
      d_output.packed_accessor64<float, 2, RestrictPtrTraits>(),
      unique_d.packed_accessor64<float, 2, RestrictPtrTraits>());
  
  return unique_d;
}


void Fused_Extra_Efficient_TT_bag_backward_sgd_cuda(
    int32_t batch_cnt,
    int32_t table_length,
    int32_t feature_dim,
    float learning_rate,
    const Tensor index,
    const Tensor rowidx,
    const std::vector<int>& tt_p_shapes, //[i1,i2,i3]
    const std::vector<int>& tt_q_shapes, //[j1,j2,j3]
    const std::vector<int>& tt_ranks, //[1,r1,r2,1]
    const Tensor tensor_p_shapes, //[i1,i2,i3]
    const Tensor tensor_q_shapes, //[j1,j2,j3]
    const Tensor tensor_ranks, //[1,r1,r2,1]
    Tensor d_reduced_output,
    std::vector<Tensor>& tt_cores,
    Tensor sorted_idx,
    Tensor sorted_key
)
{
    at::cuda::OptionalCUDAGuard device_guard;
    device_guard.set_index(d_reduced_output.get_device());

    TORCH_CHECK(index.numel() == rowidx.numel());

    int32_t T = 3;  //3
    int32_t N = batch_cnt;
    int32_t B = d_reduced_output.size(0);  // batch size
    int32_t unique_num = sorted_idx.size(0);
    

    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;

    /*
    auto d_output = at::empty({batch_cnt, feature_dim}, tt_cores[0].options());
    auto unique_d = at::zeros({sorted_idx.size(0), d_output.size(1)}, tt_cores[0].options());

    // *************** generate d_output *******************
    int32_t threads = (N > 256 ? 256 : 32);
    int32_t num_blocks = (N + threads - 1) / threads;
    generate_d_output_kernel<<<num_blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
      N,
      B,
      feature_dim,
      rowidx.data_ptr<int64_t>(),
      d_reduced_output.data_ptr<float>(),
      d_output.data_ptr<float>()
    );

    // std::cout << "d_output numel()" << d_output.numel() << std::endl;
    // std::cout << "unique_d numel()" << unique_d.numel() << std::endl;

    // *************** geneate unique_d *******************
    threads = (N > 256 ? 256 : 32);
    num_blocks = (N + threads - 1) / threads;
    compute_unique_gradient<<<num_blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
      batch_cnt,
      feature_dim,
      (const int64_t*)sorted_key.data_ptr(),
      d_output.packed_accessor64<float, 2, RestrictPtrTraits>(),
      unique_d.packed_accessor64<float, 2, RestrictPtrTraits>()
    );
    */

  auto unique_d = aggregate_gradients_v1(
    N,
    B,
    feature_dim,
    d_reduced_output,
    rowidx,
    sorted_idx,
    sorted_key
  );


  // auto unique_d =
  //     aggregate_gradients(nnz, num_uni, tr_output_dim, feature_dim, rowidx, unique_keys, reduced_d);

    //===================================================================================================================
    std::vector<Tensor> tr_tt_cores;
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[0].size(1)}, tt_cores[0].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[1].size(1)}, tt_cores[1].options()));
    tr_tt_cores.push_back(at::empty({unique_num, tt_cores[2].size(1)}, tt_cores[2].options()));

    // for (auto t : tr_tt_cores) {
    //   std::cout << "tr_tt_cores numel()" << t.numel() << std::endl;
    // }

    std::vector<int32_t> m(T - 1);
    std::vector<int32_t> n(T - 1);
    std::vector<int32_t> k(T - 1);
    float alpha = 1.0;
    float beta = 0.0;
    int32_t m_ = tt_q_shapes[0]; 
    for (int32_t t = 0; t < T - 1; ++t) {
        m[t] = m_; //m[0]=j1 m[1]=j1*j2
        k[t] = tt_ranks[t + 1]; //k[0]=r1 k[1]=r2
        n[t] = tt_q_shapes[t + 1] * tt_ranks[t + 2]; //n[0]=j2*r2 n[1]=j3
        m_ = m_ * tt_q_shapes[t + 1];
    }

    std::vector<Tensor> tr;

    int64_t tr_size = tt_q_shapes[0] * tt_ranks[1];
    for (int32_t t = 0; t < T - 2; ++t) {
        tr_size = tr_size * tt_q_shapes[t + 1] * tt_ranks[t + 2] / tt_ranks[t + 1];
        tr.push_back(at::empty({unique_num, tr_size}, tt_cores[0].options()));
    }
    // for (auto t : tr) {
    //   std::cout << "tr numel()" << t.numel() << std::endl;
    // }

    auto tt_idx =
      at::empty({T * unique_num}, tt_cores[0].options().dtype(at::kInt));
    auto a_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c_ptr_tensor = at::empty(
        {(T - 2) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a_ptr = (float**)a_ptr_tensor.data_ptr<int64_t>();
    float** b_ptr = (float**)b_ptr_tensor.data_ptr<int64_t>();
    float** c_ptr = (float**)c_ptr_tensor.data_ptr<int64_t>();
    auto a0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c0_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a0_ptr = (float**)a0_ptr_tensor.data_ptr<int64_t>();
    float** b0_ptr = (float**)b0_ptr_tensor.data_ptr<int64_t>();
    float** c0_ptr = (float**)c0_ptr_tensor.data_ptr<int64_t>();
    auto a1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto b1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    auto c1_ptr_tensor = at::empty(
        {(T - 1) * unique_num}, tt_cores[0].options().dtype(at::kLong));
    float** a1_ptr = (float**)a1_ptr_tensor.data_ptr<int64_t>();
    float** b1_ptr = (float**)b1_ptr_tensor.data_ptr<int64_t>();
    float** c1_ptr = (float**)c1_ptr_tensor.data_ptr<int64_t>();

    threads = (unique_num > 256 ? 256 : 32);
    num_blocks = (unique_num + threads - 1) / threads;

    Extra_Eff_prepare_batch_gemm_pointers_3_core_backward<<<
        num_blocks,
        threads,
        0,
        c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        (const int64_t*)sorted_idx.data_ptr(),
        (const int64_t*)tensor_p_shapes.data_ptr(),

        tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),

        tr_tt_cores[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[1].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr_tt_cores[2].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tr[0].packed_accessor64<float, 2, RestrictPtrTraits>(),
        unique_d.packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_idx.data_ptr<int32_t>(),
        a_ptr,
        b_ptr,
        c_ptr,
        a0_ptr,
        b0_ptr,
        c0_ptr,
        a1_ptr,
        b1_ptr,
        c1_ptr
    );
    cuda_gemm_batched_fp32_fp32(
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n[0],
        m[0],
        k[0],
        &alpha,
        (void**)&(a_ptr[0]),
        n[0],
        (void**)&(b_ptr[0]),
        k[0],
        &beta,
        (void**)&(c_ptr[0]),
        n[0],
        unique_num);
  

// //======================================================
    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[1],
      k[1],
      m[1],
      &alpha,
      (void**)&(b0_ptr[unique_num]),
      n[1],
      (void**)&(a0_ptr[unique_num]),
      k[1],
      &beta,
      (void**)&(c0_ptr[unique_num]),
      n[1],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[1],
      m[1],
      n[1],
      &alpha,
      (void**)&(b1_ptr[unique_num]),
      n[1],
      (void**)&(a1_ptr[unique_num]),
      n[1],
      &beta,
      (void**)&(c1_ptr[unique_num]),
      k[1],
      unique_num
    );

    // std::cout << "b: " << sorted_idx << tt_cores[0].mean() <<  std::endl;

    

//=========================================================

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      n[0],
      k[0],
      m[0],
      &alpha,
      (void**)&(b0_ptr[0]),
      n[0],
      (void**)&(a0_ptr[0]),
      k[0],
      &beta,
      (void**)&(c0_ptr[0]),
      n[0],
      unique_num
    );

    cuda_gemm_batched_fp32_fp32(
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      k[0],
      m[0],
      n[0],
      &alpha,
      (void**)&(b1_ptr[0]),
      n[0],
      (void**)&(a1_ptr[0]),
      n[0],
      &beta,
      (void**)&(c1_ptr[0]),
      k[0],
      unique_num
    );
  
    // print(tr_tt_cores[1]);
    // print(tr_tt_cores[0]);

  // printf("%.21f, %.21f, %.21f ", tt_cores[0].mean().item().toFloat(),
  // tt_cores[1].mean().item().toFloat(),
  //  tt_cores[2].mean().item().toFloat());
  // printf("%.21f, %.21f, %.21f",  tr_tt_cores[0].mean().item().toFloat(), 
  // tr_tt_cores[1].mean().item().toFloat(), 
  //  tr_tt_cores[2].mean().item().toFloat());
//=========================================================
  for (int32_t t = 0; t < T; ++t) {
      int32_t D_0 = tt_cores[t].size(1);
      int32_t tx_0 = std::min(1024, D_0);
      int32_t ty_0 = 1024 / tx_0;

      // Tensor tt_idx_ = tt_idx.narrow(0, t * unique_num, unique_num);
      // std::cout << tt_cores[t].sizes() << std::endl;
      // std::cout << "(" << tt_idx_.max().item() << ", " << tt_idx_.min().item()  << ")" << std::endl;
      // TORCH_CHECK(tt_idx_.max().item().toInt() < tt_cores[t].size(0), std::to_string(tt_idx_.max().item().toInt()) + " " + std::to_string(tt_cores[t].size(0)));

      extra_fused_update_tt_cores_sgd_kernel<<<
      div_round_up(unique_num, ty_0),
      dim3(tx_0, ty_0),
      0,
      c10::cuda::getCurrentCUDAStream()>>>(
        unique_num,
        D_0,
        learning_rate,
        &(tt_idx.data_ptr<int32_t>()[t * unique_num]),
        tr_tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>(),
        tt_cores[t].packed_accessor64<float, 2, RestrictPtrTraits>()
      );
  }
  // printf(", %.21f, %.21f, %.21f \n", tt_cores[0].mean().item().toFloat(),
  // tt_cores[1].mean().item().toFloat(),
  //  tt_cores[2].mean().item().toFloat());
  return;
}

